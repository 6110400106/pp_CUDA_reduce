
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 2048

//Interleave addressing kernel version
__global__ void interleaved_reduce(int* d_in, int* d_out) {
	int i = threadIdx.x;
	//int M = N/2;
	__shared__ int sB[N];		//shared-block memory
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("d_in[%d]: %d\n", id, d_in[id]);
	sB[i] = d_in[id];
	__syncthreads();
	// slight consideration in using
       	// s = s<<1 instead of s = s*2
	// for aesthetic purposes
	for(int s = 1; s < blockDim.x; s = s*2) {
		/*if(i < M) {
			printf("stride: %d and thread %d is active \n", s, i);
			//d_in[(2*s)*i] = d_in[(2*s)*i] + d_in[(2*s)*i+s];	
		}*/
		int index = 2 * s * id;
		if(index < blockDim.x) {
			//printf("stride: %d and thread %d is active \n", s, i);
			sB[index] += sB[index+s];
		}
		__syncthreads();
		//M = M/2;
	}
	if(i == 0)
		//d_out[0] = d_in[0];
		d_out[blockIdx.x] = sB[0];
}

//Contiguous addressing kernel version
__global__ void contiguous_reduce(int* d_in, int* d_out) {
	/*
	//What teacher taught me
	int i = threadIdx.x;
        int M = N/2;
        for(int s = M; s > 0; s=s>>1) {	// s=>>1 means right shift one bit
					// or means s = s/2
                if(i < M) {
                        printf("stride: %d and thread %d is active \n", s, i);
                        d_in[i] = d_in[i] + d_in[i+s];
                }
                M = M/2;
        }
        if(i == 0)
                d_out[0] = d_in[0];
	*/

	// What I implemented myself
	// parallel sum by using per-block shared memory
	int i = threadIdx.x;
	int id =  blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int sB[N];           //shared-block memory
	sB[i] = d_in[id];
        __syncthreads();
	//s=>>1 means right shift one bit
        // or means s = s/2
        for(int s = blockDim.x/2; s > 0; s=s>>1) {
                if(i < s) {
                        //printf("stride: %d and thread %d is active \n", s, i);
                        sB[i] += sB[i+s];
                }
            	__syncthreads();
        }
        if(i == 0)
                d_out[blockIdx.x] = sB[0];

}


int main() {
	int h_in[N];
	int h_out;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for(int i = 0; i < N; i++) {
		h_in[i] = i+1;	
	}
	
	int *d_in, *d_out;

	//Part 1: Memory transfer from host to device
	hipMalloc((void**) &d_in, N*sizeof(int));
	hipMalloc((void**) &d_out, sizeof(int));

	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);

	//Part 2: Execute kernel

	//Timed interleaved_reduce function
	/*cudaEventRecord(start);
	interleaved_reduce<<<1, 1024>>>(d_in, d_out);
	cudaEventRecord(stop);*/
	
	//Timed contiguos_reduce function
	hipEventRecord(start);
	contiguous_reduce<<<1, 1024>>>(d_in, d_out);
	hipEventRecord(stop);

	//Part 3: Memory transfer from device to host
	hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipFree(d_in);
	hipFree(d_out);

	printf("Output: %d\n", h_out);
	printf("Time used: %f milliseconds\n", milliseconds);

	return -1;
}
